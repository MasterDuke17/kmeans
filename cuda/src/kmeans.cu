#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "point.h"
#include "configurations.h"

__global__ void km_group_by_cluster(Point* points, Centroid* centroids,
        int num_centroids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = 0;

    float minor_distance = -1;

    for (i = 0; i < num_centroids; i++) {
        float diff = km_distance(&points[idx], &centroids[i]);

        // se a diferenca for menor que a menor distancia existente,
        // ou minor distance nao tiver sido inicializada
        if (minor_distance > diff || minor_distance == -1.0) {
            minor_distance = diff;
            points[idx].cluster = i;
        }
    }
}

__global__ void km_sum_points_cluter(Point* points, Centroid* centroids,
        int num_centroids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < num_centroids; i++) {
        if (points[idx].cluster == i) {
            atomicAdd(&centroids[i].x_sum, points[idx].x);
            atomicAdd(&centroids[i].y_sum, points[idx].y);
            atomicAdd(&centroids[i].num_points, 1);
        }
    }
}

__global__ void km_update_centroids(Centroid* centroids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (centroids[idx].num_points > 0) {
        centroids[idx].x = centroids[idx].x_sum / centroids[idx].num_points;
        centroids[idx].y = centroids[idx].y_sum / centroids[idx].num_points;
    }
}

__global__ void km_points_compare(Point* p1, Point* p2, int num_points,
        int *result)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_points) {
        // if any points has its cluster different, changes the result variable
        if (p1[idx].cluster != p2[idx].cluster) {
            *result = 0;
        }
    }
}

/**
* Copy points from host memory to device memory
*/
void copy_points_to_kernel(Point* h_points, Point* d_points, int array_size) {
    hipMalloc((void **) &d_points, sizeof(Point) * array_size);
    hipMemcpy(d_points, h_points, sizeof(Point) * array_size, hipMemcpyHostToDevice);
}

/**
* Copy centroids from host memory to device memory.
*/
void copy_centroids_to_kernel(Centroid* h_centroids, Centroid* d_centroids, int array_size) {
    hipMalloc((void **) &d_centroids, sizeof(Centroid) * array_size);
    hipMemcpy(d_centroids, h_centroids, sizeof(Centroid) * array_size, hipMemcpyHostToDevice);
}

/**
* Executes the k-mean algorithm.
*/
void km_execute(Point* h_points, Centroid* h_centroids, int num_points,
        int num_centroids)
{
    int continue_iterations = 1;
    int iterations;
    Point* d_points;
    Centroid* d_centroids;

    copy_points_to_kernel(h_points, d_points, num_points);
    copy_centroids_to_kernel(h_centroids, d_centroids, num_centroids);

    while (continue_iterations) {
        iterations++;

        // TODO: call kernel here! 

        if (REPOSITORY_SPECIFICATION == 1) {
            // in repository specifications, 
            // we just want know if number of 
            // iterations is equals NUMBER_OF_ITERATIONS
            if (iterations == NUMBER_OF_ITERATIONS) {
                continue_iterations = 0;
            }
        } else {
            // TODO: TEST centroids of last iteration equals actual centroids
            continue_iterations = 0; // set 1 here, just for pre implementation
        }
    }

    hipFree(d_points);
    hipFree(d_centroids);
}
