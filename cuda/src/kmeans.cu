#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "point.h"
#include "configurations.h"

__global__ void km_group_by_cluster(Point* points, Centroid* centroids,
        int num_centroids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = 0;

    float minor_distance = -1;

    for (i = 0; i < num_centroids; i++) {
        float diff = km_distance(&points[idx], &centroids[i]);

        // se a diferenca for menor que a menor distancia existente,
        // ou minor distance nao tiver sido inicializada
        if (minor_distance > diff || minor_distance == -1.0) {
            minor_distance = diff;
            points[idx].cluster = i;
        }
    }
}

__global__ void km_sum_points_cluter(Point* points, Centroid* centroids,
        int num_centroids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < num_centroids; i++) {
        if (points[idx].cluster == i) {
            atomicAdd(&centroids[i].x_sum, points[idx].x);
            atomicAdd(&centroids[i].y_sum, points[idx].y);
            atomicAdd(&centroids[i].num_points, 1);
        }
    }
}

__global__ void km_update_centroids(Centroid* centroids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (centroids[idx].num_points > 0) {
        centroids[idx].x = centroids[idx].x_sum / centroids[idx].num_points;
        centroids[idx].y = centroids[idx].y_sum / centroids[idx].num_points;
    }
}

__global__ void km_points_compare(Point* p1, Point* p2, int num_points,
        int *result)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_points) {
        // if any points has its cluster different, changes the result variable
        if (p1[idx].cluster != p2[idx].cluster) {
            *result = 0;
        }
    }
}

__global__ void km_points_copy(Point* p_dest, Point* p_src, int num_points)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_points) {
        p_dest[idx] = p_src[idx];
    }
}

/**
* Copy points from host memory to device memory
*/
void copy_points_to_kernel(Point* h_points, Point* d_points, int array_size) {
    hipMalloc((void **) &d_points, sizeof(Point) * array_size);
    hipMemcpy(d_points, h_points, sizeof(Point) * array_size, hipMemcpyHostToDevice);
}

/**
* Copy centroids from host memory to device memory.
*/
void copy_centroids_to_kernel(Centroid* h_centroids, Centroid* d_centroids, int array_size) {
    hipMalloc((void **) &d_centroids, sizeof(Centroid) * array_size);
    hipMemcpy(d_centroids, h_centroids, sizeof(Centroid) * array_size, hipMemcpyHostToDevice);
}

/**
* Executes the k-mean algorithm.
*/
void km_execute(Point* h_points, Centroid* h_centroids, int num_points,
        int num_centroids)
{
    int iterations = 0;
    Point* d_points;
    Point* d_points_old;
    Centroid* d_centroids;
    int h_res = 1;
    int *d_res;

    hipMalloc((void**)&d_res, sizeof(int));

    hipMalloc((void**)&d_points_old, sizeof(Point) * num_points);

    copy_points_to_kernel(h_points, d_points, num_points);
    copy_centroids_to_kernel(h_centroids, d_centroids, num_centroids);

    for (;;) {
        km_group_by_cluster<<<ceil(num_points/10), 10>>>(d_points, d_centroids,
                num_centroids);

        km_sum_points_cluter<<<ceil(num_points/10), 10>>>(d_points, d_centroids,
                num_centroids);

        km_update_centroids<<<ceil(num_centroids/10), 10>>>(d_centroids);

        if (REPOSITORY_SPECIFICATION == 1) {
            // in repository specifications, 
            // we just want know if number of 
            // iterations is equals NUMBER_OF_ITERATIONS
            if (iterations == NUMBER_OF_ITERATIONS) {
                break;
            }
        } else if (iterations > 0) {
            hipMemcpy(d_res, &h_res , sizeof(int), hipMemcpyHostToDevice);
            km_points_compare<<<ceil(num_points/10), 10>>>(d_points, d_points_old,
                    num_points, d_res);

            hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);

            // if h_rest == 1 the two vector of points are equal and the kmeans iterations
            // has completed all work
            if (h_res == 1)
                break;
        }
        km_points_copy<<<ceil(num_points/10), 10>>>(d_points_old, d_points,
            num_points);
        iterations++;
    }

    hipFree(d_points);
    hipFree(d_centroids);
}
