#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "point.h"

__global__ void km_group_by_cluster(Point* points, Centroid* centroids,
        int num_centroids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = 0;

    float minor_distance = -1;

    for (i = 0; i < num_centroids; i++) {
        float diff = km_distance(&points[idx], &centroids[i]);

        // se a diferenca for menor que a menor distancia existente,
        // ou minor distance nao tiver sido inicializada
        if (minor_distance > diff || minor_distance == -1.0) {
            minor_distance = diff;
            points[idx].cluster = i;
        }
    }
}

__global__ void km_sum_points_cluter(Point* points, Centroid* centroids,
        int num_centroids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < num_centroids; i++) {
        if (points[idx].cluster == i) {
            atomicAdd(&centroids[i].x_sum, points[idx].x);
            atomicAdd(&centroids[i].y_sum, points[idx].y);
            atomicAdd(&centroids[i].num_points, 1);
        }
    }
}

__global__ void km_update_centroids(Centroid* centroids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (centroids[idx].num_points > 0) {
        centroids[idx].x = centroids[idx].x_sum / centroids[idx].num_points;
        centroids[idx].y = centroids[idx].y_sum / centroids[idx].num_points;
    }
}

__global__ void km_points_compare(Point* p1, Point* p2, int num_points,
        int *result)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_points) {
        // if any points has its cluster different, changes the result variable
        if (p1[idx].cluster != p2[idx].cluster) {
            *result = 0;
        }
    }
}

void km_execute(Point* h_points, Centroid* h_centroids, int num_points,
        int num_centroids)
{

}
